
#include "hip/hip_runtime.h"

#include <stdlib.h> 
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <string>
#include <sstream>
#include <hiprand/hiprand_kernel.h>

__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje); printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla = getchar();
		exit(-1);
	}
}

/*Funci�n que gneera un n�mero aleatorio, comprendido entre 0 y el n-1 filas o columas que tenga*/
__host__ void generate_random (int *result, int elements, int max) {
	std::srand(static_cast<int>(time(0)));
	int i = 0;
	bool repeat;
	do {
		repeat = false;
		result[i] = static_cast<int>(rand() % max);
		for (int j = 0; j < i; ++j) {
			repeat |= (result[i] == result[j]);
		}
		if (!repeat) {
			++i;
		}
	} while (i < elements);
}

__host__ void printTablero(float *tablero, int n_filas, int n_columnas) {
	//Resultado
	for (int i = 0; i < n_filas; i++) {
		for (int j = 0; j < n_columnas; j++) {
			std::cout << tablero[i*n_columnas + j] << ", ";
		}
		std::cout << std::endl;
	}
}

__global__ void fillMatrix(float *tablero, int *positions, int max_elements, int n_positions, int max_random) {
	int id = threadIdx.x;
	bool set = false;
	if (id < max_elements) {
		for (int i = 0; i < n_positions; ++i) {
			if (id == (positions[i])) {
				hiprandState state;
				hiprand_init((unsigned long long)clock() + id, 0, 0, &state);
				switch (static_cast<int>(hiprand(&state) % max_random)) {
				case 0:
					tablero[id] = 2;
					break;
				case 1:
					tablero[id] = 4;
					break;
				case 2:
					tablero[id] = 8;
					break;
				}
				set = true;
			}
		}
		if (!set) {
			tablero[id] = static_cast<float>(0);
		}
	}
}

__global__ void moverDeDerechaAIzquierda(float *tablero, int size) {
	int id = threadIdx.x * size;
	int i;
	bool hay_hueco;
	int ultimo_hueco;
	float ultima_ficha;
	int ultima_ficha_posicion;
	hay_hueco = tablero[id] == 0;
	if (hay_hueco) {
		ultimo_hueco = id;
		ultima_ficha = 0;
		ultima_ficha_posicion = id;
	}
	else {
		ultima_ficha = tablero[id];
		ultima_ficha_posicion = id;
		ultimo_hueco = id;
	}
	for (int e = 1; e < size; ++e) {
		i = id + e;
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion + 1;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco <= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					++ultimo_hueco;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}


int main(int argc, char **argv) {
	float *tablero_h; //tablero de juego en el host 
	float *tablero_d; //tablero de juego en el device
	int n_filas; //numero de filas
	int n_columnas; //numro de  columnas
	int n_elementos;  //numero de elementos de la matriz (nc*nf)
	size_t size_elementos;
	int elementos_iniciales; //Nivel de juego, 8 o 15 semillas.
	char modo_ejecucion; //modo de ejecuci�n, autom�tico o manual 

	int *random_h; //vector que almacena posicion x 
	int *random_d; //vector donde se copian los puntos en el device

	if (argc < 4) {
		std::cout << "Modo de ejecucion [ a | m]" << std::endl;
		std::cin >> modo_ejecucion;
		std::cout << "Cuantos elementos iniciales quiere [ 1 = 8 | 2 = 15 ]" << std::endl;
		std::cin >> elementos_iniciales;
		std::cout << "Introduzca el numero de filas del tablero" << std::endl;
		std::cin >> n_filas;
		std::cout << "Introduzca el numero de columnas del tablero" << std::endl;
		std::cin >> n_columnas;
	} else {
		n_filas = std::atoi(argv[3]);
		n_columnas = std::atoi(argv[2]);
		modo_ejecucion = static_cast<char>(std::atoi(argv[1]));
		elementos_iniciales = std::atoi(argv[0]);
	}
	if (n_filas < 4) {
		std::cout << "Filas insuficientes" << std::endl;
		exit(-1);
	}
	if (n_columnas < 4) {
		std::cout << "Columnas insuficientes" << std::endl;
		exit(-2);
	}
	if ((modo_ejecucion != 'a') && (modo_ejecucion != 'm')) {
		std::cout << "Modo de ejecuci�n incorrecto" << std::endl;
		exit(-3);
	}
	if (elementos_iniciales < 0) {
		std::cout << "Elementos iniciales insuficiente" << std::endl;
		exit(-4);
	}
	switch (elementos_iniciales) {
	case 0: {
		elementos_iniciales = 2;
	} break;
	case 1: {
		elementos_iniciales = 8;
	} break;
	case 2: {
		elementos_iniciales = 15;
	} break;
	}
	
	n_elementos = n_filas * n_columnas;
	size_elementos = sizeof(float) * n_elementos;
	int n_elementos_pow2 = static_cast<char>(pow(2,ceil(log2(n_elementos))));

	//incializamos las posiciones iniciales aleatoriamente
	random_h = (int*) malloc(sizeof(int) * elementos_iniciales);
	generate_random(random_h, elementos_iniciales, n_elementos);
	hipMalloc((void **)&random_d, sizeof(int)*elementos_iniciales);
	hipMemcpy(random_d, random_h, sizeof(int)*elementos_iniciales, hipMemcpyHostToDevice);

	//iniciamos el tablero
	tablero_h = (float*)malloc(size_elementos);
	hipMalloc((void **)&tablero_d, size_elementos);
	fillMatrix <<<1, n_elementos_pow2, 1>>> (tablero_d, random_d, n_elementos, elementos_iniciales, static_cast<int>(floor(elementos_iniciales/3)));
	check_CUDA_Error("FILL_MATRIX");
	hipMemcpy(tablero_h, tablero_d, size_elementos, hipMemcpyDeviceToHost);
	printTablero(tablero_h, n_filas, n_columnas);
	std::cout << "---------------------" << std::endl;

	moverDeDerechaAIzquierda <<<1, n_filas, 1 >>> (tablero_d, n_columnas);
	check_CUDA_Error("MOVER");
	hipMemcpy(tablero_h, tablero_d, size_elementos, hipMemcpyDeviceToHost);
	printTablero(tablero_h, n_filas, n_columnas);
	hipFree(random_d);

	getchar(); //se cierra la ventana si no pongo esto. 
	free(tablero_h);
	hipFree(tablero_d);
	return(0);
}
